
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU (void) {
	
	printf("Hello from GPU\n");
}

int main(void) {
	
	helloFromGPU<<<1,10>>>();
	hipDeviceReset();
	return 0;
}
